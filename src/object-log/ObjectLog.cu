#include "hip/hip_runtime.h"

#include "ArmorPlate.h"
#include "ObjectLog.h"

ObjectLog::ObjectLog() : _plates(std::vector<ArmorPlate>()), _idAssign(0), _outputLog(fopen("ObjectLog.txt", "w")) {}

// 
int ObjectLog::boxesInput(std::vector<BoundingBox> boxList, time_t currTime)
{
    if (boxList.empty())
    {
        return -1;
    }

    // for(int i = 0; i < boxList.size(); i++) {
    //     printf("box %d: (%f, %f, %f)\n", i+1,  boxList[i].getXCenter(), boxList[i].getYCenter(), boxList[i].getDepthVal());
    // }

    if (_plates.empty())
    {
        for (int i = 0; i < boxList.size(); i++)
        {
            BoundingBox box = boxList[i];
            if (!sizeCheck(&box))
                continue;
            if ((box.getXCenter() < 0) || (box.getYCenter() < 0) || (box.getDepthVal() < 0) || (box.getHeight() < 0) || (box.getWidth() < 0))
            {
                return -1; // maybe change to just a continue?
            }
            ArmorPlate *newPlate = new ArmorPlate(_idAssign);
            newPlate->setLastTime(currTime);
            newPlate->setPosition(box.getPosition());
            // newPlate.addArmorPlate(newPlate, currTime);
            _plates.push_back(*newPlate);
            _idAssign++;
        }
        
    }
    else
    {
        for (int i = 0; i < boxList.size(); i++)
        {
            BoundingBox box = boxList[i];
            if (!sizeCheck(&box))
                continue;

            if ((box.getXCenter() < 0) || (box.getYCenter() < 0) || (box.getDepthVal() < 0) || (box.getHeight() < 0) || (box.getWidth() < 0))
            {
                return -1;
            }
            int assoc = assign_plate(&box, _plates);
            // printf("ASSOC %d\n", assoc);
            if (assoc == -1)
            {
                ArmorPlate *newAP = new ArmorPlate(_idAssign);
                newAP->setLastTime(currTime);
                newAP->setPosition(std::tuple<float, float, float>(box.getXCenter(), box.getYCenter(), box.getDepthVal()));
                if (_plates.size() < 9)
                {
                    _plates.push_back(*newAP);
                    _idAssign++;
                }
                else
                {
                    std::cout << "need space" << std::endl;
                }
            }
            else if (assoc == -2)
            {
                std::cout << "panic" << std::endl;
            }
            else if (assoc == -3)
            {
                std::cout << "out of range" << std::endl;
            }
            else
            {
                _plates[assoc].setLastTime(currTime);
                _plates[assoc].setIsActive(true);
                _plates[assoc].setPosition(box.getPosition());
                // _idAssign++;
            }
        }

        for (int i = 0; i < _plates.size(); i++)
        {
            ArmorPlate plate = _plates[i];
            if (currTime - plate.getLastTime() > KILL_THRESHOLD)
            {
                kill_plate(plate.getId());
                // plate.timeBuffer++;
                // if (plate.timeBuffer == kill_threshold)
                // {
                //     kill_plate(plate.getID()); // originally kill_plate(i) but I think that is wrong
                // }
                
            }
        }
    }
    for(int i = 0; i < _plates.size(); i++) {
        std::tuple<float, float, float> pos = _plates[i].getPosition();
        // printf("plate %d: (%f, %f, %f)\n", _plates[i].getId(), std::get<0>(pos), std::get<1>(pos), std::get<2>(pos));
    }
    return 0;
}

// Basic check to see if a bounding box meets the basic requirements (size does matter)
bool ObjectLog::sizeCheck(BoundingBox *box)
{
    return (box->getHeight() * box->getWidth()) >= MIN_AREA;
}

int ObjectLog::assign_plate(BoundingBox *box, std::vector<ArmorPlate> plates)
{
    if (box == NULL || &plates == NULL)
        return -2;

    std::tuple<float, float, float> position = box->getPosition();
    // printf("BOX: (%f, %f, %f)\n", std::get<0>(position), std::get<1>(position), std::get<2>(position));
    float shortest_dist = std::numeric_limits<float>::max();
    int shortest_plate = -1;
    if (((std::get<0>(position) + MARGIN_OF_ERR) > MAX_X) || ((std::get<1>(position) + MARGIN_OF_ERR) > MAX_Y) || ((std::get<2>(position) + MARGIN_OF_ERR) > MAX_Z) 
            || ((std::get<0>(position) - MARGIN_OF_ERR) < MIN_X) || ((std::get<1>(position) - MARGIN_OF_ERR) < MIN_Y) || ((std::get<2>(position) - MARGIN_OF_ERR) < MIN_Z))
    {
        return -3;
    }

    for (int i = 0; i < plates.size(); i++)
    {
        float dist = get_distance(position, plates[i].getPosition());
        if (dist < shortest_dist)
        {   
            shortest_plate = i;
            shortest_dist = dist;
            // printf("shortest distance: %f\n", shortest_dist);
        }
    }

    float full_mog = sqrt(3 * pow(MARGIN_OF_ERR, 2)); // full mog represents the margin of error extended to 3d space
    if (shortest_dist > full_mog)
    {
        // printf("RETURN -1, %f %f\n", shortest_dist, full_mog);
        return -1;
    }
    // printf("SHORTEST PLATE: %d, SHORTEST DIST: %f\n", shortest_plate, shortest_dist);
    return shortest_plate;
}

void ObjectLog::kill_all()
{
    // for (int i = 0; i < _plates.size(); i++)
    // {
    //     _plates[i].writeToHistory(_outputLog)
    // }
    _plates.clear();
    fclose(_outputLog);
    return;
}

std::vector<ArmorPlate> ObjectLog::get_plates()
{
    return _plates;
}

void ObjectLog::kill_plate(int id)
{
    for (int i = 0; i < _plates.size(); i++)
    {
        if (_plates[i].getId() == id)
        {
            // _plates[i].writeToHistory(_outputLog);
            _plates.erase(_plates.begin() + i);
            break;
        }
    }
}

// Distance formula (basically Pythagorean theorem in 3D space)
float ObjectLog::get_distance(std::tuple<float, float, float> p1, std::tuple<float, float, float> p2)
{
    return sqrt(pow((std::get<0>(p1) - std::get<0>(p2)), 2) + pow((std::get<1>(p1) - std::get<1>(p2)), 2) + pow((std::get<2>(p1) - std::get<2>(p2)), 2));
}