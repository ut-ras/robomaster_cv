#include "hip/hip_runtime.h"

#include "ArmorPlate.h"
#include "ObjectLog.h"

ObjectLog::ObjectLog() : _plates(std::vector<ArmorPlate>()), _idAssign(0), _outputLog(fopen("ObjectLog.txt", "w")) {}

// 
int ObjectLog::boxesInput(std::vector<BoundingBox> boxList, time_t currTime)
{
    if (boxList.empty())
    {
        return -1;
    }

    if (_plates.empty())
    {
        for (int i = 0; i < boxList.size(); i++)
        {
            BoundingBox box = boxList[i];
            if (!sizeCheck(&box))
                continue;
            if ((box.getXCenter() < 0) || (box.getYCenter() < 0) || (box.getDepthVal() < 0) || (box.getHeight() < 0) || (box.getWidth() < 0))
            {
                return -1; // maybe change to just a continue?
            }
            ArmorPlate *newPlate = new ArmorPlate(_idAssign);
            newPlate->setLastTime(currTime);
            newPlate->setPosition({box.getXCenter(), box.getYCenter(), box.getDepthVal()});
            // newPlate.addArmorPlate(newPlate, currTime);
            _plates.push_back(*newPlate);
            _idAssign++;
        }
    }
    else
    {
        for (int i = 0; i < boxList.size(); i++)
        {
            BoundingBox box = boxList[i];
            // std::cout << box << std::endl; //C++ does not have the same printing properties as Python so prints may be useless
            if (!sizeCheck(&box))
                continue;

            if ((box.getXCenter() < 0) || (box.getYCenter() < 0) || (box.getDepthVal() < 0) || (box.getHeight() < 0) || (box.getWidth() < 0))
            {
                return -1;
            }
            int assoc = assign_plate(&box, _plates);
            ArmorPlate *newAP = new ArmorPlate(_idAssign);
            newAP->setLastTime(currTime);
            newAP->setPosition(std::tuple<float, float, float>(box.getXCenter(), box.getYCenter(), box.getDepthVal()));
            if (assoc == -1)
            {
                if (_plates.size() < 9)
                {
                    _plates.push_back(*newAP);
                    _idAssign++;
                }
                else
                {
                    std::cout << "need space" << std::endl;
                }
            }
            else if (assoc == -2)
            {
                std::cout << "panic" << std::endl;
            }
            else if (assoc == -3)
            {
                std::cout << "out of range" << std::endl;
            }
            else
            {
                ArmorPlate assocPlate = _plates[assoc];
                // assocPlate.addArmorPlate(newAP, currTime);
                // assocPlate.timeBuffer = 0;
                assocPlate.setLastTime(currTime);
                assocPlate.setIsActive(true);
                _idAssign++;
            }
        }

        for (int i = 0; i < _plates.size(); i++)
        {
            ArmorPlate plate = _plates[i];
            if (currTime - plate.getLastTime() > KILL_THRESHOLD)
            {
                kill_plate(plate.getId());
                // plate.timeBuffer++;
                // if (plate.timeBuffer == kill_threshold)
                // {
                //     kill_plate(plate.getID()); // originally kill_plate(i) but I think that is wrong
                // }
                
            }
        }
    }
}

// Basic check to see if a bounding box meets the basic requirements (size does matter)
bool ObjectLog::sizeCheck(BoundingBox *box)
{
    return (box->getHeight() * box->getWidth()) >= MIN_AREA;
}

int ObjectLog::assign_plate(BoundingBox *box, std::vector<ArmorPlate> plates)
{
    if (box == NULL || &plates == NULL)
        return -2;

    std::tuple<float, float, float> position = box->getPosition();
    float shortest_dist = std::numeric_limits<float>::max();
    int shortest_plate = -1;
    float error = MARGIN_OF_ERR;
    if (((std::get<0>(position) + MARGIN_OF_ERR) > MAX_X) || ((std::get<1>(position) + MARGIN_OF_ERR) > MAX_Y) || ((std::get<2>(position) + MARGIN_OF_ERR) > MAX_Z) 
            || ((std::get<0>(position) - MARGIN_OF_ERR) < MIN_X) || ((std::get<1>(position) - MARGIN_OF_ERR) < MIN_Y) || ((std::get<2>(position) - MARGIN_OF_ERR) < MIN_Z))
    {
        return -3;
    }

    for (int i = 0; i < plates.size(); i++)
    {
        double dist = get_distance(position, plates[i].getPosition());
        if (dist < shortest_dist)
        {
            shortest_plate = i;
            shortest_dist = dist;
        }
    }

    if (shortest_dist > MARGIN_OF_ERR)
    {
        return -1;
    }

    return shortest_plate;
}

void ObjectLog::kill_all()
{
    // for (int i = 0; i < _plates.size(); i++)
    // {
    //     _plates[i].writeToHistory(_outputLog)
    // }
    _plates.clear();
    fclose(_outputLog);
    return;
}

std::vector<ArmorPlate> ObjectLog::get_plates()
{
    return _plates;
}

void ObjectLog::kill_plate(int id)
{
    for (int i = 0; i < _plates.size(); i++)
    {
        if (_plates[i].getId() == id)
        {
            // _plates[i].writeToHistory(_outputLog);
            _plates.erase(_plates.begin() + i);
            break;
        }
    }
}

// Distance formula (basically Pythagorean theorem in 3D space)
double get_distance(std::tuple<double, double, double> p1, std::tuple<double, double, double> p2)
{
    return sqrt(pow((std::get<0>(p1) - std::get<0>(p2)), 2) + pow((std::get<1>(p1) - std::get<1>(p2)), 2) + pow((std::get<2>(p1) - std::get<2>(p2)), 2));
}