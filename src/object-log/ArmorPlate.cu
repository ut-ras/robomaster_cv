#include <stdio.h>
#include "ArmorPlate.h"
#include "hipblas.h"
#include <iostream>


// ! WE MIGHT NEED TO CHANGE ALL THE floatS HERE TO FLOATS
ArmorPlate::ArmorPlate(int id)
    : _id(id),
      _position(std::tuple<float, float, float>(0, 0, 0)),
      _velocity(std::tuple<float, float, float>(0, 0, 0)),
      _acceleration(std::tuple<float, float, float>(0, 0, 0)),
      _boundingBox(BoundingBox()),
      _isActive(true),
      _seenThisIteration(false),
      _next_position(std::tuple<float, float, float>(0, 0, 0)),
      _lastTime(time(0)),
      _associatedBoxes(std::vector<BoundingBox>())
// _kalmanFilter(new KalmanFilter())
{
    /*
     * Initializes the armor plate
     * Rundown of the fields of the object:
     * position, holds the position of the bounding box in a x, y, z system (camera relative)
     * velocity, last velocity of the target
     * acceleration, last acceleration of the target
     * boundingbox, boundingbox object
     * id, id of the armor plate (for debugging purposes)
     * activity, boolean on the plate on if it is currently alive
     */
}

int ArmorPlate::getId()
{
    return _id;
}

std::tuple<float, float, float> ArmorPlate::getPosition()
{
    return _position;
}

std::tuple<float, float, float> ArmorPlate::getVelocity()
{
    return _velocity;
}

std::tuple<float, float, float> ArmorPlate::getAcceleration()
{
    return _acceleration;
}

bool ArmorPlate::getIsActive()
{
    return _isActive;
}

bool ArmorPlate::getSeenThisIteration()
{
    return _seenThisIteration;
}

std::tuple<float, float, float> ArmorPlate::getNextPosition()
{
    return _next_position;
}

time_t ArmorPlate::getLastTime()
{
    return _lastTime;
}

BoundingBox ArmorPlate::getBoundingBox()
{
    return _boundingBox;
}

std::vector<BoundingBox> ArmorPlate::getAssociatedBoxes()
{
    return _associatedBoxes;
}

void ArmorPlate::setId(int id)
{
    ArmorPlate::_id = id;
}

void ArmorPlate::setPosition(std::tuple<float, float, float> position)
{
    ArmorPlate::_position = position;
}

void ArmorPlate::setVelocity(std::tuple<float, float, float> velocity)
{
    ArmorPlate::_velocity = velocity;
}

void ArmorPlate::setAcceleration(std::tuple<float, float, float> acceleration)
{
    ArmorPlate::_acceleration = acceleration;
}

void ArmorPlate::setIsActive(bool isActive)
{
    ArmorPlate::_isActive = isActive;
}

void ArmorPlate::setSeenThisIteration(bool seenThisIteration)
{
    ArmorPlate::_seenThisIteration = seenThisIteration;
}

void ArmorPlate::setNextPosition(std::tuple<float, float, float> next_position)
{
    ArmorPlate::_next_position = next_position;
}

void ArmorPlate::setLastTime(time_t lastTime)
{
    ArmorPlate::_lastTime = lastTime;
}

// void setBoundingBox(BoundingBox boundingBox) {}
// void setKalmanFilter(KalmanFilter kalmanFilter) {}

void ArmorPlate::updatePositionVelAcc()
{
    /*
     * Updates the position, velocity, and acceleration of the armor plate
     * Uses the kalman filter to do so
     */

    // ! TODO implement this
    // get the predicted position from kalman filter
    // get predicted vel and acc from the kalman filter
    // set the position, vel, and acc to the predicted values
}

void ArmorPlate::predictPosition(time_t currentTime)
{
    /*
     * Predicts the position of the armor plate at the current time
     * Uses the kalman filter to do so
     */
    // get the predicted position from kalman filter
    // set the position to the predicted value
    time_t timeDiff = currentTime - ArmorPlate::_lastTime;
    // kinematics ut + 0.5at^2
    std::tuple<float, float, float> deltaVel = std::tuple<float, float, float>(0, 0, 0);
    float *velocity, *acceleration, *deltaVel_h;

    // TODO we need to get velocity and acceleration from the kalman filter
    velocity = (float *)malloc(3 * sizeof(float));
    acceleration = (float *)malloc(3 * sizeof(float));
    deltaVel_h = (float *)malloc(3 * sizeof(float));
    
    velocity[0] = (std::get<0>(ArmorPlate::_velocity));
    velocity[1] = (std::get<1>(ArmorPlate::_velocity));
    velocity[2] = (std::get<2>(ArmorPlate::_velocity));

    acceleration[0] = (std::get<0>(ArmorPlate::_acceleration));
    acceleration[1] = (std::get<1>(ArmorPlate::_acceleration));
    acceleration[2] = (std::get<2>(ArmorPlate::_acceleration));

    deltaVel_h[0] = (std::get<0>(deltaVel));
    deltaVel_h[1] = (std::get<1>(deltaVel));
    deltaVel_h[2] = (std::get<2>(deltaVel));

    hipblasHandle_t handle;
    hipblasStatus_t status = hipblasCreate(&handle);

    float *velocity_d, *acceleration_d, *deltaVel_d;
    hipMalloc((void **)&velocity_d, 3 * sizeof(float));
    hipMalloc((void **)&acceleration_d, 3 * sizeof(float));
    hipMalloc((void **)&deltaVel_d, 3 * sizeof(float));

    hipblasSetVector(3, sizeof(float), velocity, 1, velocity_d, 1);
    hipblasSetVector(3, sizeof(float), acceleration, 1, acceleration_d, 1);
    hipblasSetVector(3, sizeof(float), deltaVel_h, 1, deltaVel_d, 1);

    float alpha = 0.5 * timeDiff * timeDiff;
    
    hipblasSaxpy_64(handle, 3, &alpha, acceleration_d, 1, deltaVel_d, 1);

    alpha = timeDiff;
    hipblasSaxpy_64(handle, 3, &alpha, velocity_d, 1, deltaVel_d, 1);

    hipblasGetVector(3, sizeof(float), deltaVel_d, 1, deltaVel_h, 1);
    
    ArmorPlate::_velocity = deltaVel;
    setNextPosition(std::tuple<float, float, float>(deltaVel_h[0], deltaVel_h[1], deltaVel_h[2]));
}

int experimentCUDA(int n)
{
    int deviceCount;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    hipblasHandle_t handle;
    hipblasStatus_t status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        std::cout << "CUBLAS initialization failed!" << std::endl;
        return 1;
    }

    // * Vector pointers for device and host
    float *d_A, *d_B, *h_A, *h_B, *h_C;

    // * Allocate memory for host vectors
    h_A = (float *)malloc(n * sizeof(float));
    h_B = (float *)malloc(n * sizeof(float));
    h_C = (float *)malloc(n * sizeof(float));

    // * Allocate memory for device vectors
    hipMalloc((void **)&d_A, n * sizeof(float));
    hipMalloc((void **)&d_B, n * sizeof(float));

    // * Initialize host vectors
    for (int i = 0; i < n; i++)
    {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }

    // * Copy host vectors to device
    hipblasSetVector(n, sizeof(float), h_A, 1, d_A, 1);
    hipblasSetVector(n, sizeof(float), h_B, 1, d_B, 1);

    // TODO look up cudaStreams because cudaSetVectorAsync can be used

    // * Perform vector addition
    float alpha = 1.0;
   

    time_t start = time(0);
    std::cout << "Starting GPU" << start << std::endl;
    hipblasSaxpy(handle, n, &alpha, d_A, 1, d_B, 1);
    time_t end = time(0);
    std::cout << "Finished GPU" << end << std::endl;
    std::cout << "GPU Time: " << end - start << std::endl;

    // * Copy result back to host
    hipblasGetVector(n, sizeof(float), d_B, 1, h_C, 1);

    // * Perform vector addition on CPU
    start = time(0);
    std::cout << "Starting CPU" << start << std::endl;
    for (int i = 0; i < n; i++)
    {
        h_C[i] = h_A[i] + h_B[i];
    }
    end = time(0);
    std::cout << "Finished CPU" << end << std::endl;
    std::cout << "CPU Time: " << end - start << std::endl;

    // * Print result
    // for(int i = 0; i < n; i++) {
    //     std::cout << h_A[i] << " + " << h_B[i] << " = " << h_C[i] << std::endl;
    // }

    // * Free device memory
    hipFree(d_A);
    hipFree(d_B);

    // * Free host memory
    free(h_A);
    free(h_B);
    free(h_C);
    std::cout << "Success" << std::endl;
    hipblasDestroy(handle);
    return 0;
}

int main()
{

    // * Size of vector
    int n = 500000000;
    int step = 10000;
    for (int i = 0; i < 100; i++)
    {
        std::cout<< "n: " << n << std::endl;
        experimentCUDA(n);
        n += step;
    }
    return 0;
}