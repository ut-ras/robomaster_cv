#include "Kalman.h"
#include <stdio.h>
#include "hipblas.h"
#include <stdio.h>
#include <iostream>

void storeColumnMajor(float src[][VECTOR_SIZE], float *dest) {
    float *dest_copy = dest;
    for(int i = 0; i < VECTOR_SIZE; i++){
        // columns
        for(int j = 0; j < VECTOR_SIZE; j++) {
            // rows
            *dest_copy++ = src[j][i]; 
        }
    }
}
Kalman::Kalman(float deltaTime)
{

    _error = hipGetDeviceCount(&_device_count);
    _status = hipblasCreate(&_handle);
    if (_status != HIPBLAS_STATUS_SUCCESS)
    {
        std::cout << "CUBLAS initialization failed!" << std::endl;
        return;
    }

    float time_factor = 0.5f * deltaTime * deltaTime;
    float _state_transition_matrix_init[][VECTOR_SIZE] = {
        {1.0, 0.0, 0.0, deltaTime, 0.0, 0.0, time_factor, 0, 0},
        {0.0, 1.0, 0.0, 0.0, deltaTime, 0.0, 0.0, time_factor, 0},
        {0.0, 0.0, 1.0, 0.0, 0.0, deltaTime, 0.0, 0.0, time_factor},
        {0.0, 0.0, 0.0, 1.0, 0.0, 0.0, deltaTime, 0.0, 0.0},
        {0.0, 0.0, 0.0, 0.0, 1.0, 0.0, 0.0, deltaTime, 0.0},
        {0.0, 0.0, 0.0, 0.0, 0.0, 1.0, 0.0, 0.0, deltaTime},
        {0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 1.0, 0.0, 0.0},
        {0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 1.0, 0.0},
        {0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 1.0},
    };
    _state_transition_matrix = (float *) malloc(VECTOR_SIZE * VECTOR_SIZE * sizeof(float*));
    storeColumnMajor(_state_transition_matrix_init, _state_transition_matrix);

    // for(int i = 0; i < VECTOR_SIZE * VECTOR_SIZE; i++) {
    //     std::cout<<_state_transition_matrix[i]<<" "<<std::endl;
    // }
};

void Kalman::updateStateTransitionMatrix(float *matrix, float deltaTime)
{
    float _state_transition_matrix_init[][VECTOR_SIZE] = {
        {1.0, 0.0, 0.0, deltaTime, 0.0, 0.0, TIME_FACTOR, 0, 0},
        {0.0, 1.0, 0.0, 0.0, deltaTime, 0.0, 0.0, TIME_FACTOR, 0},
        {0.0, 0.0, 1.0, 0.0, 0.0, deltaTime, 0.0, 0.0, TIME_FACTOR},
        {0.0, 0.0, 0.0, 1.0, 0.0, 0.0, deltaTime, 0.0, 0.0},
        {0.0, 0.0, 0.0, 0.0, 1.0, 0.0, 0.0, deltaTime, 0.0},
        {0.0, 0.0, 0.0, 0.0, 0.0, 1.0, 0.0, 0.0, deltaTime},
        {0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 1.0, 0.0, 0.0},
        {0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 1.0, 0.0},
        {0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 1.0},
    };
    
    storeColumnMajor(_state_transition_matrix_init, matrix);

}

int main()
{
    Kalman *k = new Kalman(1.2);
}
