#include "hip/hip_runtime.h"

// #include "object-log/ArmorPlate.h"
// #include "object-log/ObjectLog.h"
#include "ArmorPlate.h"
#include "ObjectLog.h"
#include <unistd.h>
#include <chrono>
#include <memory>

ObjectLog::ObjectLog() : _plates(std::vector<ArmorPlate>()), _idAssign(0), _outputLog(fopen("ObjectLog.txt", "w")) {}

//
int ObjectLog::boxesInput(std::vector<BoundingBox> boxList, uint64_t currTime)
{
    if (boxList.empty())
    {
        return -1;
    }

    // for(int i = 0; i < boxList.size(); i++) {
    //     printf("box %d: (%f, %f, %f)\n", i+1,  boxList[i].getXCenter(), boxList[i].getYCenter(), boxList[i].getDepthVal());
    // }

    // ! Remove later *******
    for (int i = 0; i < boxList.size(); i++)
    {
        printf("box %d: (%f, %f, %f)\n", i + 1, boxList[i].getXCenter(), boxList[i].getYCenter(), boxList[i].getDepthVal());
    }

    if (_plates.empty())
    {
        for (int i = 0; i < boxList.size(); i++)
        {
            BoundingBox box = boxList[i];
            if (!sizeCheck(&box)) {
                printf("Failed size check");
                continue;
            }
            if ((box.getXCenter() < 0) || (box.getYCenter() < 0) || (box.getDepthVal() < 0) || (box.getHeight() < 0) || (box.getWidth() < 0))
            {
                printf("Negative values");
                continue;
            }
            auto newPlate = std::shared_ptr<ArmorPlate>(new ArmorPlate(_idAssign));
            newPlate->setLastTime(currTime);
            newPlate->setPosition(box.getPosition(), currTime);
            printf("box position: (%f, %f, %f)\n", std::get<0>(box.getPosition()), std::get<1>(box.getPosition()), std::get<2>(box.getPosition()));
            printf("newPlate position: (%f, %f, %f)\n", std::get<0>(newPlate->getPosition()), std::get<1>(newPlate->getPosition()), std::get<2>(newPlate->getPosition()));
            // newPlate.addArmorPlate(newPlate, currTime);
            _plates.push_back(*newPlate);
            _idAssign++;
        }
    }
    else
    {
        for (int i = 0; i < boxList.size(); i++)
        {
            BoundingBox box = boxList[i];
            printf("Processing box: (%f, %f, %f)\n", box.getXCenter(), box.getYCenter(), box.getDepthVal());
            if (!sizeCheck(&box))
                continue;

            if ((box.getXCenter() < 0) || (box.getYCenter() < 0) || (box.getDepthVal() < 0) || (box.getHeight() < 0) || (box.getWidth() < 0))
            {
                return -1;
            }
            int assoc = assign_plate(&box, _plates);
            printf("ASSOC %d\n", assoc);
            if (assoc == -1)
            {
                if (_plates.size() < 9)
                {
                    auto newAP = std::shared_ptr<ArmorPlate>(new ArmorPlate(_idAssign));
                    newAP->setLastTime(currTime);
                    newAP->setPosition(std::tuple<float, float, float>(box.getXCenter(), box.getYCenter(), box.getDepthVal()), currTime);
                    printf("New Armor Plate position: (%f, %f, %f)\n", std::get<0>(newAP->getPosition()), std::get<1>(newAP->getPosition()), std::get<2>(newAP->getPosition()));
                    _plates.push_back(*newAP);
                    _idAssign++;
                }
                else
                {
                    std::cout << "need space" << std::endl;
                }
            }
            else if (assoc == -2)
            {
                std::cout << "panic" << std::endl;
            }
            else if (assoc == -3)
            {
                std::cout << "out of range" << std::endl;
            }
            else
            {
                // We made an association
                printf("Previous plate velocity: (%f, %f, %f)\n", std::get<0>(_plates[assoc].getVelocity()), std::get<1>(_plates[assoc].getVelocity()), std::get<2>(_plates[assoc].getVelocity()));
                printf("Previous plate position: (%f, %f, %f)\n", std::get<0>(_plates[assoc].getPosition()), std::get<1>(_plates[assoc].getPosition()), std::get<2>(_plates[assoc].getPosition()));
                printf("Previous plate acceleration: (%f, %f, %f)\n", std::get<0>(_plates[assoc].getAcceleration()), std::get<1>(_plates[assoc].getAcceleration()), std::get<2>(_plates[assoc].getAcceleration()));
                printf("Previous plate last time: %ld\n", _plates[assoc].getLastTime());

                _plates[assoc].setIsActive(true);
                _plates[assoc].updatePositionVelAcc();
                _plates[assoc].setPosition(box.getPosition(), currTime);
                _plates[assoc].setLastTime(currTime);
                printf("ASSOC %d\n", assoc);
                printf("POS: (%f, %f, %f)\n", std::get<0>(_plates[assoc].getPosition()), std::get<1>(_plates[assoc].getPosition()), std::get<2>(_plates[assoc].getPosition()));

                // _idAssign++;
            }
        }

        for (int i = 0; i < _plates.size(); i++)
        {
            ArmorPlate plate = _plates[i];
            printf("Current time: %ld, plate.getLastTime(): %ld\n", currTime, plate.getLastTime());
            if (currTime - plate.getLastTime() > KILL_THRESHOLD)
            {
                kill_plate(plate.getId());
                // plate.timeBuffer++;
                // if (plate.timeBuffer == kill_threshold)
                // {
                //     kill_plate(plate.getID()); // originally kill_plate(i) but I think that is wrong
                // }
            }
        }
    }
    // for(int i = 0; i < _plates.size(); i++) {
    //     std::tuple<float, float, float> pos = _plates[i].getPosition();
    // printf("plate %d: (%f, %f, %f)\n", _plates[i].getId(), std::get<0>(pos), std::get<1>(pos), std::get<2>(pos));
    // }
    return 0;
}

// a function to decide which Armor Plate to shoot at
std::vector<float> ObjectLog::getFinalArmorPlateState()
{
    int center_x = FRAME_WIDTH / 2;
    int center_y = FRAME_HEIGHT / 2;

    float distance = std::numeric_limits<float>::max();
    int best_index = -1;
    std::cout << "_plate.size(): " << _plates.size() << std::endl;
    for (size_t i = 0; i < _plates.size(); i++)
    {
        float plate_distance = get_distance(_plates[i].getPosition(), std::tuple<float, float, float>(center_x, center_y, std::get<2>(_plates[i].getPosition())));
        if (plate_distance < distance)
        {
            best_index = i;
        }
    }

    // get the position velocity and accelaration from the plate
    // pack it into a float vector
    std::vector<float> plate_state;
    if (best_index >= 0) {
        plate_state.push_back(std::get<0>(_plates[best_index].getPosition()));
        plate_state.push_back(std::get<1>(_plates[best_index].getPosition()));
        plate_state.push_back(std::get<2>(_plates[best_index].getPosition()));
        plate_state.push_back(std::get<0>(_plates[best_index].getVelocity()));
        plate_state.push_back(std::get<1>(_plates[best_index].getVelocity()));
        plate_state.push_back(std::get<2>(_plates[best_index].getVelocity()));
        plate_state.push_back(std::get<0>(_plates[best_index].getAcceleration()));
        plate_state.push_back(std::get<1>(_plates[best_index].getAcceleration()));
        plate_state.push_back(std::get<2>(_plates[best_index].getAcceleration()));
    }
    return plate_state;
}

// Basic check to see if a bounding box meets the basic requirements (size does matter)
bool ObjectLog::sizeCheck(BoundingBox *box)
{
    return (box->getHeight() * box->getWidth()) >= MIN_AREA;
}

int ObjectLog::assign_plate(BoundingBox *box, std::vector<ArmorPlate> plates)
{
    if (box == NULL || &plates == NULL)
        return -2;

    std::tuple<float, float, float> position = box->getPosition();
    printf("BOX: (%f, %f, %f)\n", std::get<0>(position), std::get<1>(position), std::get<2>(position));
    float shortest_dist = std::numeric_limits<float>::max();
    int shortest_plate = -1;
    printf("Condition 1: %d\n", ((std::get<0>(position) + MARGIN_OF_ERR) > MAX_X));
    printf("Condition 2: %d\n", ((std::get<1>(position) + MARGIN_OF_ERR) > MAX_Y));
    printf("Condition 3: %d\n", ((std::get<2>(position) + MARGIN_OF_ERR) > MAX_Z));
    printf("Condition 4: %d\n", ((std::get<0>(position) - MARGIN_OF_ERR) < MIN_X));
    printf("Condition 5: %d\n", ((std::get<1>(position) - MARGIN_OF_ERR) < MIN_Y));
    printf("Condition 6: %d\n", ((std::get<2>(position) - MARGIN_OF_ERR) < MIN_Z));
    printf("Min check X: %f\n", (std::get<0>(position) - MARGIN_OF_ERR));
    printf("Min check Y: %f\n", (std::get<1>(position) - MARGIN_OF_ERR));
    printf("Min check Z: %f\n", (std::get<2>(position) - MARGIN_OF_ERR));
    // if (((std::get<0>(position) + MARGIN_OF_ERR) > MAX_X) || ((std::get<1>(position) + MARGIN_OF_ERR) > MAX_Y) || ((std::get<2>(position) + MARGIN_OF_ERR) > MAX_Z) || ((std::get<0>(position) - MARGIN_OF_ERR) < MIN_X) || ((std::get<1>(position) - MARGIN_OF_ERR) < MIN_Y) || ((std::get<2>(position) - MARGIN_OF_ERR) < MIN_Z))
    // {
    //     return -3;
    // }

    for (int i = 0; i < plates.size(); i++)
    {
        float dist = get_distance(position, plates[i].getPosition());
        printf("Plate %d: (%f, %f, %f)\n", i, std::get<0>(plates[i].getPosition()), std::get<1>(plates[i].getPosition()), std::get<2>(plates[i].getPosition()));
        printf("Position: (%f, %f, %f)\n", std::get<0>(position), std::get<1>(position), std::get<2>(position));
        printf("Distance: %f\n", dist);
        if (dist < shortest_dist)
        {
            shortest_plate = i;
            shortest_dist = dist;
        }
    }

    float full_mog = sqrt(3 * pow(MARGIN_OF_ERR, 2)); // full mog represents the margin of error extended to 3d space
    printf("full mog: %f\n", full_mog);
    printf("shortest dist: %f\n", shortest_dist);
    if (shortest_dist > full_mog)
    {
        return -1;
    }
    return shortest_plate;
}

void ObjectLog::kill_all()
{
    // for (int i = 0; i < _plates.size(); i++)
    // {
    //     _plates[i].writeToHistory(_outputLog)
    // }
    _plates.clear();
    fclose(_outputLog);
    return;
}

std::vector<ArmorPlate> ObjectLog::get_plates()
{
    return _plates;
}

void ObjectLog::kill_plate(int id)
{
    for (int i = 0; i < _plates.size(); i++)
    {
        if (_plates[i].getId() == id)
        {
            // _plates[i].writeToHistory(_outputLog);
            // delete &(_plates[i]);
            _plates.erase(_plates.begin() + i);
            break;
        }
    }
}

// Distance formula (basically Pythagorean theorem in 3D space)
float ObjectLog::get_distance(std::tuple<float, float, float> p1, std::tuple<float, float, float> p2)
{
    return sqrt(pow((std::get<0>(p1) - std::get<0>(p2)), 2) + pow((std::get<1>(p1) - std::get<1>(p2)), 2) + pow((std::get<2>(p1) - std::get<2>(p2)), 2));
}

// int main()
// {
//     std::vector<BoundingBox> boxList;
//     BoundingBox box = BoundingBox();
//     float x = 50.0;
//     float y = 50.0;
//     float z = 50.0;
//     ObjectLog *log = new ObjectLog();
//     for (int i = 0; i < 10; i++)
//     {
//         box.setXCenter(x);
//         box.setYCenter(y);
//         box.setDepthVal(z);
//         box.setWidth(10.0);
//         box.setHeight(10.0);
//         boxList.push_back(box);
//         time_t currTime;
//         time(&currTime);
//         log->boxesInput(boxList, double(currTime));
//         printf("Time: %ld\n", currTime);
//         std::vector<float> plate_state = log->getFinalArmorPlateState();
//         printf("target: \n");
//         for (int i = 0; i < plate_state.size(); i++)
//         {
//             printf("%f\n", plate_state[i]);
//         }

//         // sleep for 0.01 seconds
//         x += 0.02;
//         y += 0.02;
//         z += 0.01;
//         boxList = std::vector<BoundingBox>();
//         sleep(1);
//     }
//     return 0;
// }