#include "hip/hip_runtime.h"
// #include "object-log/Kalman.h"
#include "Kalman.h"
#include <stdio.h>
#include "hipblas.h"
#include <stdio.h>
#include <iostream>
#include <time.h>

/*
 * @param src: 2D array to be stored in column major format
 * @param dest: pointer to the destination array
 * @return void
 */
void Kalman::storeColumnMajor(float src[][VECTOR_SIZE], float *dest)
{
    float *dest_copy = dest;
    for (int i = 0; i < VECTOR_SIZE; i++)
    {
        // columns
        for (int j = 0; j < VECTOR_SIZE; j++)
        {
            // rows
            *dest_copy++ = src[j][i];
        }
    }
}

Kalman::Kalman(float deltaTime)
{

    _error = hipGetDeviceCount(&_device_count);
    _status = hipblasCreate(&_handle);
    if (_status != HIPBLAS_STATUS_SUCCESS)
    {
        std::cout << "CUBLAS initialization failed!" << std::endl;
        return;
    }

    _state_n_1 = (float *)calloc(sizeof(float *), VECTOR_SIZE);
    _state_n = (float *)calloc(sizeof(float *), VECTOR_SIZE);

    float time_factor = 0.5f * deltaTime * deltaTime;
    float _state_transition_matrix_init[][VECTOR_SIZE] = {
        {1.0, 0.0, 0.0, deltaTime, 0.0, 0.0, time_factor, 0, 0},
        {0.0, 1.0, 0.0, 0.0, deltaTime, 0.0, 0.0, time_factor, 0},
        {0.0, 0.0, 1.0, 0.0, 0.0, deltaTime, 0.0, 0.0, time_factor},
        {0.0, 0.0, 0.0, 1.0, 0.0, 0.0, deltaTime, 0.0, 0.0},
        {0.0, 0.0, 0.0, 0.0, 1.0, 0.0, 0.0, deltaTime, 0.0},
        {0.0, 0.0, 0.0, 0.0, 0.0, 1.0, 0.0, 0.0, deltaTime},
        {0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 1.0, 0.0, 0.0},
        {0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 1.0, 0.0},
        {0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 1.0},
    };
    _state_transition_matrix = (float *)malloc(VECTOR_SIZE * VECTOR_SIZE * sizeof(float *));
    storeColumnMajor(_state_transition_matrix_init, _state_transition_matrix);
};

void Kalman::updateStateTransitionMatrix(float *matrix, float deltaTime)
{
    float _state_transition_matrix_init[][VECTOR_SIZE] = {
        {1.0, 0.0, 0.0, deltaTime, 0.0, 0.0, TIME_FACTOR, 0, 0},
        {0.0, 1.0, 0.0, 0.0, deltaTime, 0.0, 0.0, TIME_FACTOR, 0},
        {0.0, 0.0, 1.0, 0.0, 0.0, deltaTime, 0.0, 0.0, TIME_FACTOR},
        {0.0, 0.0, 0.0, 1.0, 0.0, 0.0, deltaTime, 0.0, 0.0},
        {0.0, 0.0, 0.0, 0.0, 1.0, 0.0, 0.0, deltaTime, 0.0},
        {0.0, 0.0, 0.0, 0.0, 0.0, 1.0, 0.0, 0.0, deltaTime},
        {0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 1.0, 0.0, 0.0},
        {0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 1.0, 0.0},
        {0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 1.0},
    };

    // stores the newly initialized matrix in column major format
    storeColumnMajor(_state_transition_matrix_init, matrix);
}

void Kalman::set_state_n(float *position, float *velocity, float *acceleration)
{
    memcpy(_state_n, position, STATE_SIZE * sizeof(float));
    memcpy(_state_n + STATE_SIZE, velocity, STATE_SIZE * sizeof(float));
    memcpy(_state_n + 2 * STATE_SIZE, acceleration, STATE_SIZE * sizeof(float));
}

void Kalman::predict_state_n_1()
{
    // device pointers for all the vectors and matrices
    float *d_state_transition_matrix, *d_state_n, *d_state_n_1;

    hipMalloc((void **)&d_state_transition_matrix, VECTOR_SIZE * VECTOR_SIZE * sizeof(float));
    hipMalloc((void **)&d_state_n, VECTOR_SIZE * sizeof(float));
    hipMalloc((void **)&d_state_n_1, VECTOR_SIZE * sizeof(float));

    hipblasStatus_t copy_status;
    copy_status = hipblasSetMatrix(VECTOR_SIZE, VECTOR_SIZE, sizeof(float), _state_transition_matrix, VECTOR_SIZE, d_state_transition_matrix, VECTOR_SIZE);
    if (copy_status != HIPBLAS_STATUS_SUCCESS)
    {
        std::cout << "Error copying state transition matrix to device" << std::endl;
        return;
    }

    copy_status = hipblasSetVector(VECTOR_SIZE, sizeof(float), _state_n, 1, d_state_n, 1);
    if (copy_status != HIPBLAS_STATUS_SUCCESS)
    {
        std::cout << "Error copying state_n to device" << std::endl;
        return;
    }

    // perform matrix multiplication
    float alpha = 1.0f;
    float beta = 0.0f;
    hipblasSgemv(_handle, HIPBLAS_OP_N, VECTOR_SIZE, VECTOR_SIZE, &alpha, d_state_transition_matrix, VECTOR_SIZE, d_state_n, 1, &beta, d_state_n_1, 1);

    // copy the result back to host
    copy_status = hipblasGetVector(VECTOR_SIZE, sizeof(float), d_state_n_1, 1, _state_n_1, 1);
    if (copy_status != HIPBLAS_STATUS_SUCCESS)
    {
        std::cout << "Error copying state_n_1 to host" << std::endl;
        return;
    }
}

/*
 * @brief Updates the state transition matrix based on deltaTime
*/
void Kalman::setDeltaTime(float deltaTime) {
    updateStateTransitionMatrix(_state_transition_matrix, deltaTime);
}

float *Kalman::get_state_n_1()
{
    return _state_n_1;
}

void Kalman::update_state_n()
{
    memcpy(_state_n, _state_n_1, VECTOR_SIZE * sizeof(float));
}

float *Kalman::get_state_n()
{
    return _state_n;
}

void Kalman::set_velocity(float *velocity)
{
    memcpy(_state_n + STATE_SIZE, velocity, STATE_SIZE * sizeof(float));
}

void Kalman::set_acceleration(float *acceleration)
{
    memcpy(_state_n + 2 * STATE_SIZE, acceleration, STATE_SIZE * sizeof(float));
}

// int main()
// {
//     /*
//      * TESTING
//      */
//     float DELTA_TIME = 1.2;
//     Kalman *k = new Kalman(DELTA_TIME);

//     float position[] = {1.0, 2.0, 3.0};
//     float velocity[] = {4.0, 5.0, 6.0};
//     float acceleration[] = {7.0, 8.0, 9.0};

//     time_t start = time(0);
//     std::cout << "Starting time: " << start << std::endl;
//     for (int i = 0; i < 20; i++)
//     {
//         k->set_state_n(position, velocity, acceleration);
//         k->predict_state_n_1();
//         float *output = k->get_state_n_1();
//         k->update_state_n();
//         output = k->get_state_n();
//         for (int i = 0; i < VECTOR_SIZE; i++)
//         {
//             std::cout << output[i] << std::endl;
//         }
//         position[0] = output[0];
//         position[1] = output[1];
//         position[2] = output[2];
//         velocity[0] = output[3];
//         velocity[1] = output[4];
//         velocity[2] = output[5];
//         acceleration[0] = output[6];
//         acceleration[1] = output[7];
//         acceleration[2] = output[8];
//         std::cout<<"----------iteration ended---------\n";
//     }
//     time_t end = time(0);
//     std::cout << "Ending time: " << end << std::endl;
//     std::cout << "Time taken: " << end - start << std::endl;
//     return 0;
// }
