#include "hip/hip_runtime.h"
// #include "object-log/BoundingBox.h"
#include "BoundingBox.h"

BoundingBox::BoundingBox(float xCenter, float yCenter, float depthVal, float width, float height)
{
    _xCenter = xCenter;
    _yCenter = yCenter;
    _depthVal = depthVal;
    _width = width;
    _height = height;
    _position = std::tuple<float, float, float>((xCenter - (width/2)), yCenter - (height/2), depthVal);
    _timeStamp = time(0);
    
}

BoundingBox::BoundingBox(){
    _xCenter = 0;
    _yCenter = 0;
    _depthVal = 0;
    _width = 0;
    _height = 0;
    _position = std::tuple<float, float, float>(0, 0, 0);
    _timeStamp = time(0);
}

void BoundingBox::setXCenter(float xCenter){
    _xCenter = xCenter;
}

void BoundingBox::setYCenter(float yCenter){
    _yCenter = yCenter;
}

void BoundingBox::setDepthVal(float depthVal){
    _depthVal = depthVal;
}

void BoundingBox::setWidth(float width){
    _width = width;
}

void BoundingBox::setHeight(float height){
    _height = height;
}

void BoundingBox::setPosition(std::tuple<float, float, float> position){
    _position = position;
}

void BoundingBox::setTimeStamp(time_t timeStamp){
    _timeStamp = timeStamp;
}

float BoundingBox::getXCenter(){
    return _xCenter;
}

float BoundingBox::getYCenter(){
    return _yCenter;
}

float BoundingBox::getDepthVal(){
    return _depthVal;
}

float BoundingBox::getWidth(){
    return _width;
}

float BoundingBox::getHeight(){
    return _height;
}

std::tuple<float, float, float> BoundingBox::getPosition(){
    std::tuple<float, float, float> position = std::make_tuple(_xCenter, _yCenter, _depthVal);
    return position;
}

time_t BoundingBox::getTimeStamp(){
    return _timeStamp;
}

// int BoundingBox::main(){
//     return 0;
// }