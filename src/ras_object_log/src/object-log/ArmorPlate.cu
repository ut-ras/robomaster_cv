#include "hip/hip_runtime.h"
#include <stdio.h>
// #include "object-log/ArmorPlate.h"
#include "ArmorPlate.h"
#include "hipblas.h"
#include <iostream>
#include <chrono>

ArmorPlate::ArmorPlate(int id)
    : _id(id),
      _position(std::tuple<float, float, float>(0, 0, 0)),
      _velocity(std::tuple<float, float, float>(0, 0, 0)),
      _acceleration(std::tuple<float, float, float>(0, 0, 0)),
      _boundingBox(BoundingBox()),
      _isActive(true),
      _seenThisIteration(false),
      _delta_position(std::tuple<float, float, float>(0, 0, 0)),
      _lastTime(time(0)),
      _associatedBoxes(std::vector<BoundingBox>()),
      _kalmanFilter(new Kalman(0.0))
{
    /*
     * Initializes the armor plate
     * Rundown of the fields of the object:
     * position, holds the position of the bounding box in a x, y, z system (camera relative)
     * velocity, last velocity of the target
     * acceleration, last acceleration of the target
     * boundingbox, boundingbox object
     * id, id of the armor plate (for debugging purposes)
     * activity, boolean on the plate on if it is currently alive
     */
}

// ArmorPlate::~ArmorPlate() {
//     delete _kalmanFilter;
// }

int ArmorPlate::getId()
{
    return _id;
}

std::tuple<float, float, float> ArmorPlate::getPosition()
{
    return _position;
}

std::tuple<float, float, float> ArmorPlate::getVelocity()
{
    return _velocity;
}

std::tuple<float, float, float> ArmorPlate::getAcceleration()
{
    return _acceleration;
}

bool ArmorPlate::getIsActive()
{
    return _isActive;
}

bool ArmorPlate::getSeenThisIteration()
{
    return _seenThisIteration;
}

std::tuple<float, float, float> ArmorPlate::getNextPosition()
{
    return _delta_position;
}

uint64_t ArmorPlate::getLastTime()
{
    return _lastTime;
}

BoundingBox ArmorPlate::getBoundingBox()
{
    return _boundingBox;
}

std::vector<BoundingBox> ArmorPlate::getAssociatedBoxes()
{
    return _associatedBoxes;
}

void ArmorPlate::setId(int id)
{
    ArmorPlate::_id = id;
}

void ArmorPlate::setPosition(std::tuple<float, float, float> position, uint64_t currentTime)
{
    printf("Current Time: %ld\n", currentTime);
    printf("Last Time: %ld\n", _lastTime);
    time_t deltaTime = currentTime - _lastTime;
    printf("Delta Time: %ld\n", deltaTime);
    std::tuple<float, float, float> new_velocity = std::tuple<float, float, float>((std::get<0>(position) - std::get<0>(ArmorPlate::_position) / double(deltaTime)),
                                                                                   (std::get<1>(position) - std::get<1>(ArmorPlate::_position) / double(deltaTime)),
                                                                                   (std::get<2>(position) - std::get<2>(ArmorPlate::_position) / double(deltaTime)));
    printf("Old Position: %f, %f, %f\n", std::get<0>(ArmorPlate::_position), std::get<1>(ArmorPlate::_position), std::get<2>(ArmorPlate::_position));
    printf("New Position: %f, %f, %f\n", std::get<0>(position), std::get<1>(position), std::get<2>(position));

    printf("New Velocity: %f, %f, %f\n", std::get<0>(new_velocity), std::get<1>(new_velocity), std::get<2>(new_velocity));
    std::tuple<float, float, float> new_accelaration = std::tuple<float, float, float>((std::get<0>(new_velocity) - std::get<0>(ArmorPlate::_velocity) / double(deltaTime)),
                                                                                       (std::get<1>(new_velocity) - std::get<1>(ArmorPlate::_velocity) / double(deltaTime)),
                                                                                       (std::get<2>(new_velocity) - std::get<2>(ArmorPlate::_velocity) / double(deltaTime)));
    printf("New Acceleration: %f, %f, %f\n", std::get<0>(new_accelaration), std::get<1>(new_accelaration), std::get<2>(new_accelaration));
    this->predictPosition(currentTime);
    // set position to the average of position and _next_position
    // ArmorPlate::_position = std::tuple<float, float, float>((std::get<0>(position) + std::get<0>(ArmorPlate::_delta_position)) / 2,
    //                                                         (std::get<1>(position) + std::get<1>(ArmorPlate::_delta_position)) / 2,
    //                                                         (std::get<2>(position) + std::get<2>(ArmorPlate::_delta_position)) / 2);
    
    if (deltaTime != 0)
    {
        this->setVelocity(new_velocity);
        this->setAcceleration(new_accelaration);
    }
    printf("Predicted delta Position: %f, %f, %f\n", std::get<0>(ArmorPlate::_delta_position), std::get<1>(ArmorPlate::_delta_position), std::get<2>(ArmorPlate::_delta_position));
    std::tuple<float, float, float> new_position = std::tuple<float, float, float>((std::get<0>(position) + std::get<0>(ArmorPlate::_delta_position)),
                                                                                   (std::get<1>(position) + std::get<1>(ArmorPlate::_delta_position)),
                                                                                   (std::get<2>(position) + std::get<2>(ArmorPlate::_delta_position)));
    printf("New Predicted Position: %f, %f, %f\n", std::get<0>(new_position), std::get<1>(new_position), std::get<2>(new_position));
    
    // set the position to the average of new_position and position
    ArmorPlate::_position = std::tuple<float, float, float>((std::get<0>(new_position) + std::get<0>(position)) / 2,
                                                            (std::get<1>(new_position) + std::get<1>(position)) / 2,
                                                            (std::get<2>(new_position) + std::get<2>(position)) / 2);
    printf("Final Position: %f, %f, %f\n", std::get<0>(ArmorPlate::_position), std::get<1>(ArmorPlate::_position), std::get<2>(ArmorPlate::_position));
}

void ArmorPlate::setVelocity(std::tuple<float, float, float> velocity)
{
    ArmorPlate::_velocity = velocity;
}

void ArmorPlate::setAcceleration(std::tuple<float, float, float> acceleration)
{
    ArmorPlate::_acceleration = acceleration;
}

void ArmorPlate::setIsActive(bool isActive)
{
    ArmorPlate::_isActive = isActive;
}

void ArmorPlate::setSeenThisIteration(bool seenThisIteration)
{
    ArmorPlate::_seenThisIteration = seenThisIteration;
}

void ArmorPlate::setNextPosition(std::tuple<float, float, float> next_position)
{
    ArmorPlate::_delta_position = next_position;
}

void ArmorPlate::setLastTime(uint64_t lastTime)
{
    ArmorPlate::_lastTime = lastTime;
}

// void setBoundingBox(BoundingBox boundingBox) {}
// void setKalmanFilter(KalmanFilter kalmanFilter) {}

// TODO (IMP) Whenever we make a callback function for the topic that listens to the position,
// TODO we need to calculate instantaneous velocity and acceleration
// TODO make two new functions for this that are similar to the functions in Kalman

void ArmorPlate::updatePositionVelAcc()
{
    /*
     * Updates the position, velocity, and acceleration of the armor plate
     * Uses the kalman filter to do so
     */

    // ! TODO implement this
    // get the predicted position from kalman filter
    // get predicted vel and acc from the kalman filter
    // set the position, vel, and acc to the predicted values
    float *position = (float *)malloc(3 * sizeof(float));
    float *velocity = (float *)malloc(3 * sizeof(float));
    float *acceleration = (float *)malloc(3 * sizeof(float));

    position[0] = (std::get<0>(ArmorPlate::_position));
    position[1] = (std::get<1>(ArmorPlate::_position));
    position[2] = (std::get<2>(ArmorPlate::_position));

    velocity[0] = (std::get<0>(ArmorPlate::_velocity));
    velocity[1] = (std::get<1>(ArmorPlate::_velocity));
    velocity[2] = (std::get<2>(ArmorPlate::_velocity));

    acceleration[0] = (std::get<0>(ArmorPlate::_acceleration));
    acceleration[1] = (std::get<1>(ArmorPlate::_acceleration));
    acceleration[2] = (std::get<2>(ArmorPlate::_acceleration));

    _kalmanFilter->set_state_n(position, velocity, acceleration);
    _kalmanFilter->predict_state_n_1();
    float *output = _kalmanFilter->get_state_n_1();
    _kalmanFilter->update_state_n();
    output = _kalmanFilter->get_state_n();
    printf("Kalman output: %f, %f, %f, %f, %f, %f, %f, %f, %f\n", output[0], output[1], output[2], output[3], output[4], output[5], output[6], output[7], output[8]);

    ArmorPlate::_position = std::tuple<float, float, float>(output[0], output[1], output[2]);
    ArmorPlate::_velocity = std::tuple<float, float, float>(output[3], output[4], output[5]);
    ArmorPlate::_acceleration = std::tuple<float, float, float>(output[6], output[7], output[8]);
}

/*
 * @brief Predicts the position of the armor plate at the current time
 * Uses the kalman filter to do so
 */
void ArmorPlate::predictPosition(uint64_t currentTime)
{

    // get the predicted position from kalman filter
    // set the position to the predicted value
    uint64_t timeDiff = currentTime - ArmorPlate::_lastTime;
    // kinematics ut + 0.5at^2
    std::tuple<float, float, float> deltaVel = std::tuple<float, float, float>(0, 0, 0);
    float *velocity, *acceleration, *deltaVel_h;

    // TODO we need to get velocity and acceleration from the kalman filter
    velocity = (float *)malloc(3 * sizeof(float));
    acceleration = (float *)malloc(3 * sizeof(float));
    deltaVel_h = (float *)malloc(3 * sizeof(float));

    velocity[0] = (std::get<0>(ArmorPlate::_velocity));
    velocity[1] = (std::get<1>(ArmorPlate::_velocity));
    velocity[2] = (std::get<2>(ArmorPlate::_velocity));

    acceleration[0] = (std::get<0>(ArmorPlate::_acceleration));
    acceleration[1] = (std::get<1>(ArmorPlate::_acceleration));
    acceleration[2] = (std::get<2>(ArmorPlate::_acceleration));

    deltaVel_h[0] = (std::get<0>(deltaVel));
    deltaVel_h[1] = (std::get<1>(deltaVel));
    deltaVel_h[2] = (std::get<2>(deltaVel));

    hipblasHandle_t handle;
    hipblasStatus_t status = hipblasCreate(&handle);

    float *velocity_d, *acceleration_d, *deltaVel_d;
    hipMalloc((void **)&velocity_d, 3 * sizeof(float));
    hipMalloc((void **)&acceleration_d, 3 * sizeof(float));
    hipMalloc((void **)&deltaVel_d, 3 * sizeof(float));

    hipblasSetVector(3, sizeof(float), velocity, 1, velocity_d, 1);
    hipblasSetVector(3, sizeof(float), acceleration, 1, acceleration_d, 1);
    hipblasSetVector(3, sizeof(float), deltaVel_h, 1, deltaVel_d, 1);

    float alpha = 0.5 * timeDiff * timeDiff;

    hipblasSaxpy(handle, 3, &alpha, acceleration_d, 1, deltaVel_d, 1);

    alpha = timeDiff;
    hipblasSaxpy(handle, 3, &alpha, velocity_d, 1, deltaVel_d, 1);

    hipblasGetVector(3, sizeof(float), deltaVel_d, 1, deltaVel_h, 1);

    ArmorPlate::_velocity = deltaVel;
    setNextPosition(std::tuple<float, float, float>(deltaVel_h[0], deltaVel_h[1], deltaVel_h[2]));
}

/*
 * @brief Updates the state transition matrix of the Kalman Filter
 */
void ArmorPlate::setDeltaTime(float deltaTime)
{
    _kalmanFilter->setDeltaTime(deltaTime);
}

int experimentCUDA(int n)
{
    int deviceCount;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    hipblasHandle_t handle;
    hipblasStatus_t status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        std::cout << "CUBLAS initialization failed!" << std::endl;
        return 1;
    }

    // * Vector pointers for device and host
    float *d_A, *d_B, *h_A, *h_B, *h_C;

    // * Allocate memory for host vectors
    h_A = (float *)malloc(n * sizeof(float));
    h_B = (float *)malloc(n * sizeof(float));
    h_C = (float *)malloc(n * sizeof(float));

    // * Allocate memory for device vectors
    hipMalloc((void **)&d_A, n * sizeof(float));
    hipMalloc((void **)&d_B, n * sizeof(float));

    // * Initialize host vectors
    for (int i = 0; i < n; i++)
    {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }

    // * Copy host vectors to device
    hipblasSetVector(n, sizeof(float), h_A, 1, d_A, 1);
    hipblasSetVector(n, sizeof(float), h_B, 1, d_B, 1);

    // TODO look up cudaStreams because cudaSetVectorAsync can be used

    // * Perform vector addition
    float alpha = 1.0;

    time_t start = time(0);
    std::cout << "Starting GPU" << start << std::endl;
    hipblasSaxpy(handle, n, &alpha, d_A, 1, d_B, 1);
    time_t end = time(0);
    std::cout << "Finished GPU" << end << std::endl;
    std::cout << "GPU Time: " << end - start << std::endl;

    // * Copy result back to host
    hipblasGetVector(n, sizeof(float), d_B, 1, h_C, 1);

    // * Perform vector addition on CPU
    start = time(0);
    std::cout << "Starting CPU" << start << std::endl;
    for (int i = 0; i < n; i++)
    {
        h_C[i] = h_A[i] + h_B[i];
    }
    end = time(0);
    std::cout << "Finished CPU" << end << std::endl;
    std::cout << "CPU Time: " << end - start << std::endl;

    // * Print result
    // for(int i = 0; i < n; i++) {
    //     std::cout << h_A[i] << " + " << h_B[i] << " = " << h_C[i] << std::endl;
    // }

    // * Free device memory
    hipFree(d_A);
    hipFree(d_B);

    // * Free host memory
    free(h_A);
    free(h_B);
    free(h_C);
    std::cout << "Success" << std::endl;
    hipblasDestroy(handle);
    return 0;
}

// int main()
// {

//     // * Size of vector
//     int n = 500000000;
//     int step = 10000;
//     for (int i = 0; i < 100; i++)
//     {
//         std::cout << "n: " << n << std::endl;
//         experimentCUDA(n);
//         n += step;
//     }
//     return 0;
// }
